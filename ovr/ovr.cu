#include "hip/hip_runtime.h"
#include "highgui.h"
#include "imgproc.h"
#include "gpu_util.h"

//const float u_distortion[4] = { 1.0, -0.22, -0.24, 0 };
// preprogrammed constants
#define UD0 1.0
#define UD1 0.22
#define UD2 0.24

__global__ void barrel_distort_ovr(float *G, float *F,
    int n_rows, int n_cols, int n_slices,
    float r_x, float r_y, float r_max, float offset_x, int right_image) {
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= n_rows || j >= n_cols) {
    return;
  }
  float x = (float)(j-r_x) / r_max + offset_x;
  float y = (float)(i-r_y) / r_max;
  // distortionScale
  float rr = x*x+y*y;
  float distortion = UD0+UD1*rr+UD2*rr*rr;
  int _i = (int)roundf(distortion*y*r_max+r_y);
  int _j = (int)roundf((distortion*x-offset_x)*r_max+r_x);
  for (int k = 0; k < n_slices; k++) {
    // since they are stored right next to each other, the slices are the separating factor (k*2+right_image)
    if (_i >= 0 && _i < n_rows && _j >= 0 && _j < n_cols) {
      G[IJK2C(i, j, k*2+right_image, n_rows, n_cols)] = F[IJK2C(_i, _j, k, n_rows, n_cols)];
    }
  }
}

gcube ovr_image(const gcube &left, const gcube &right, double offset_x) {
  // assume that the left and right images have equal dimensions
  assert(left.n_rows == right.n_rows && left.n_cols == right.n_cols);
  double mrx = left.n_cols / 2.0 * (1 + abs(offset_x));
  double mry = right.n_cols / 2.0;
  float r_max = (float)sqrt(mrx * mrx + mry * mry);
  float r_x = left.n_cols / 2.0f;
  float r_y = left.n_rows / 2.0f;
  dim3 blockSize(16, 16, 1);
  dim3 gridSize((left.n_cols-1)/16+1, (left.n_rows-1)/16+1, 1);
  gcube C(left.n_rows, left.n_cols + right.n_cols, left.n_slices, gfill::zeros);
  barrel_distort_ovr<<<gridSize, blockSize>>>(C.d_pixels, left.d_pixels,
      left.n_rows, left.n_cols, left.n_slices,
      r_x, r_y, r_max, -offset_x, 0);
  checkCudaErrors(hipGetLastError());
  barrel_distort_ovr<<<gridSize, blockSize>>>(C.d_pixels, right.d_pixels,
      right.n_rows, right.n_cols, right.n_slices,
      r_x, r_y, r_max, offset_x, 1);
  checkCudaErrors(hipGetLastError());
  return gpu_imresize2(C, 800, 1280);
}
